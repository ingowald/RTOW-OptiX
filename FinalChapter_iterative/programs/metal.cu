#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "material.h"
#include "prd.h"
#include "sampling.h"

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
/*! the per ray data we operate on */
rtDeclareVariable(PerRayData, prd,   rtPayload, );
rtDeclareVariable(rtObject,   world, , );



/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(float3, hit_rec_normal, attribute hit_rec_normal, );
rtDeclareVariable(float3, hit_rec_p, attribute hit_rec_p, );


/*! and finally - that particular material's parameters */
rtDeclareVariable(float3, albedo, , );
rtDeclareVariable(float,  fuzz,   , );


/*! the actual scatter function - in Pete's reference code, that's a
  virtual function, but since we have a different function per program
  we do not need this here */
inline __device__ bool scatter(const optix::Ray &ray_in,
                               DRand48 &rndState,
                               vec3f &scattered_origin,
                               vec3f &scattered_direction,
                               vec3f &attenuation)
{
  float3 hit_pt_world = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_rec_p);
  float3 normal_world = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, hit_rec_normal));

  float3 hrn = normal_world;
  vec3f reflected = reflect(unit_vector(ray_in.direction),normal_world);
  scattered_origin    = hit_pt_world;
  scattered_direction = (reflected+fuzz*random_in_unit_sphere(rndState));
  attenuation         = vec3f(1.f);//albedo;
  return (dot(scattered_direction, hrn) > 0.f);
}

RT_PROGRAM void closest_hit()
{
  prd.out.scatterEvent
    = scatter(ray,
              *prd.in.randState,
              prd.out.scattered_origin,
              prd.out.scattered_direction,
              prd.out.attenuation)
    ? rayGotBounced
    : rayGotCancelled;
}
