#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_world.h>
#include "prd.h"

/*! the parameters that describe each individual sphere geometry */
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float,  radius, , );

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(float3, hit_rec_normal, attribute hit_rec_normal, );
rtDeclareVariable(float3, hit_rec_p, attribute hit_rec_p, );

/*! the per ray data we operate on */
rtDeclareVariable(PerRayData, prd, rtPayload, );


// Program that performs the ray-sphere intersection
//
// note that this is here is a simple, but not necessarily most numerically
// stable ray-sphere intersection variant out there. There are more
// stable variants out there, but for now let's stick with the one that
// the reference code used.
RT_PROGRAM void hit_sphere(int pid)
{
  // See Ch. 7: Precision Improvements for Ray/Sphere Intersection
  // p.87 of Ray Tracing Gems, edited by Eric Haines and Tomas Akenine-Moller, Apress 2019.
  // http://www.realtimerendering.com/raytracinggems/
  const float3 d = ray.direction;
  const float3 f = ray.origin - center; //TODO: center assumed to be zero, we can simplify this
  const float  a = dot(d, d);
  const float  b_prime = dot(-f, d);
  const float  r = radius;  //TODO: radius assumed to be one, we can simplify this
  const float3  l = f + (b_prime/a)*d; //not sure this is actually equivalent to l in the text?
  const float  discriminant = r*r - dot(l,l);
  
  if (discriminant < 0.f) return;

  float c = dot(f,f) - r*r;
  float q = b_prime + copysignf(sqrt(a*discriminant),b_prime);

  float t0 = c/q; 
  if (t0 < ray.tmax && t0 > ray.tmin) {
    if (rtPotentialIntersection(t0)) {
      hit_rec_p = ray.origin + t0 * ray.direction;
      /*hit_rec_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,(hit_rec_p - center) / radius));*/
      hit_rec_normal = hit_rec_p; //assumes center 0, radius 1. We transform to world in a later stage.
      rtReportIntersection(0);
    }
  }

  float t1 = q/a;
  if (t1 < ray.tmax && t1 > ray.tmin) {
    if (rtPotentialIntersection(t1)) {
      hit_rec_p = ray.origin + t1 * ray.direction;
      /*hit_rec_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,(hit_rec_p - center) / radius));*/
      hit_rec_normal = hit_rec_p; //assumes center 0, radius 1. We transform to world in a later stage.
      rtReportIntersection(0);
    }
  }
}

/*! returns the bounding box of the pid'th primitive
  in this gometry. Since we only have one sphere in this 
  program (we handle multiple spheres by having a different
  geometry per sphere), the'pid' parameter is ignored */
RT_PROGRAM void get_bounds(int pid, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->m_min = center - radius;
  aabb->m_max = center + radius;
}
